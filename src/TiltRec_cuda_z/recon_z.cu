#include "hip/hip_runtime.h"
#include "recon_z.cuh"
extern hipDeviceProp_t deviceProps;

void CuBackProjectZ(Point3DF &origin, MrcStackM &projs,
                    std::vector<SimCoeff> &params,
                    MrcStackM &mrcvol, Slice &proj,
                    Volume &vol, const options &opt)
{
    int thickness = opt.thickness;
    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;

    int steplength = thickness;
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * steplength;
    size_t projsize = (size_t)projs.X() * projs.Y() * projsnum;
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR
    CuTaskDataZ cudev;
    CuMallocBPTTaskDataZ(cudev, projsnum, projs.X(), projs.Y(), steplength);
    float *originalProjsData;
    hipMallocManaged((void **)(&originalProjsData), sizeof(float) * projsize); // the data of proj is extended by steplength
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(),
               hipMemcpyHostToDevice);
    CUERR
    {
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (originalProjsData + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (originalProjsData + (size_t)projs.X() * projs.Y() * j));
    }
    CUERR
    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR

    for (int z = vol.z; z < vol.z + vol.height; z += steplength)
    {
        if (z + steplength >= vol.z + vol.height)
        { // compenstate for margin
            steplength = vol.z + vol.height - z;
            volsize = (size_t)projs.X() * projs.Y() * steplength;
            projsize = (size_t)projs.X() * projs.Y() * projsnum;
            cudev.z = steplength;
        }

        hipDeviceSynchronize();
        CUERR
        dim3 dimBlock = maxThreadsSize;
        dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                      steplength, projsnum);
        CuBackProjKernelZ<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs,
                                                  vol.data, originalProjsData, cudev.x,
                                                  cudev.y, z);
        CUERR
        hipDeviceSynchronize();
        CUERR
              if (opt.f2b)
        {
            thrust::device_ptr<float> dev_ptr(vol.data);
            float chunk_mean = thrust::reduce(dev_ptr, dev_ptr + volsize) / volsize;
            float chunk_std = thrust::transform_reduce(dev_ptr, dev_ptr + volsize, [chunk_mean] __device__(float x)
                                                       { return (x - chunk_mean) * (x - chunk_mean); }, 0.0f, thrust::plus<float>());
            chunk_std = sqrt(chunk_std / volsize);

            CufloatToByteKernel<<<(volsize / 4 + maxThreadsSize - 1) / maxThreadsSize, dimBlock>>>(vol.data, chunk_mean, chunk_std, 10, volsize);
            int j = 0;
            for (j = 0; j + 20 < thickness; j += 20)
                mrcvol.WriteBlock<unsigned char>(j, j + 20, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
            mrcvol.WriteBlock<unsigned char>(j, thickness, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
        }
        else
        {
            int j = 0;
            for (j = 0; j + 20 < thickness; j += 20)
                mrcvol.WriteBlock<float>(j, j + 20, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
            mrcvol.WriteBlock<float>(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        }
    }

    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipFree(originalProjsData);
}

void CuSIRTZ(Point3DF &origin, MrcStackM &projs, std::vector<SimCoeff> &params,
             MrcStackM &mrcvol, Slice &proj, Volume &vol,
             const options &opt)
{

     int thickness = opt.thickness;
    int iteration = opt.iteration;
    float gamma = opt.gamma;
    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;

    int batchsize = 1;
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * thickness;
    size_t oneProjsize = (size_t)projs.X() * projs.Y() * batchsize;
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR
    CuTaskDataZ cudev;
    CuMallocSIRTTaskDataZ(cudev, projs.Z(), projs.X(), projs.Y(), thickness, batchsize);
    CUERR
    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(),
               hipMemcpyHostToDevice);
    CUERR

    float *valvol, *wtvol;
    dim3 dimBlock = maxThreadsSize;
    dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  thickness, batchsize);
    dim3 dim2Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  batchsize);
    hipMallocManaged((void **)&valvol, sizeof(float) * volsize);
    hipMallocManaged((void **)&wtvol, sizeof(float) * volsize);
    CUERR
    hipMallocManaged((void **)&proj.data, sizeof(float) * (size_t)projs.X() * projs.Y() * projs.Z());
    {
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
    }

    for (int iter = 0; iter < iteration; ++iter)
    {
        hipMemset(valvol, 0, sizeof(float) * volsize);
        hipMemset(wtvol, 0, sizeof(float) * volsize);
        for (int projIdxStart = 0; projIdxStart < projsnum; projIdxStart += batchsize)
        {
            float *curProjData = proj.data + projIdxStart * projs.X() * projs.Y();
            hipDeviceSynchronize();
            printf("SIRT Iter %d on projs [%d,%d)\n", iter, projIdxStart, projIdxStart + batchsize);
            hipMemset(cudev.c, 0, sizeof(float) * oneProjsize);
            hipMemset(cudev.s, 0, sizeof(float) * oneProjsize);
            CuReprojectKernelZ<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs,
                                                       vol.data, cudev.s, cudev.c,
                                                       cudev.x, cudev.y, 0, projIdxStart);
            CUERR

            CuCalcProjectionDiffKernelZ<<<dim2Grid, dimBlock>>>(
                curProjData, cudev.s, cudev.c, cudev.x, cudev.y);
            CUERR
            CuBackProjWeightAndValueKernelZ<<<dim3Grid, dimBlock>>>(
                cudev.origin, cudev.coeffs, valvol, wtvol, cudev.s, cudev.x, cudev.y,
                0, projIdxStart);

            hipDeviceSynchronize();

            CUERR
        }

        CuUpdateVolumeByWeightsKernelZ<<<
            (volsize + maxThreadsSize - 1) / maxThreadsSize, dimBlock>>>(
            vol.data, valvol, wtvol, gamma, volsize);
        CUERR
        hipDeviceSynchronize();
        CUERR
    }
    hipDeviceSynchronize();
    CUERR
        if (opt.f2b)
    {
        thrust::device_ptr<float> dev_ptr(vol.data);
        float chunk_mean = thrust::reduce(dev_ptr, dev_ptr + volsize) / volsize;
        float chunk_std = thrust::transform_reduce(dev_ptr, dev_ptr + volsize, [chunk_mean] __device__(float x)
                                                   { return (x - chunk_mean) * (x - chunk_mean); }, 0.0f, thrust::plus<float>());
        chunk_std = sqrt(chunk_std / volsize);

        CufloatToByteKernel<<<(volsize / 4 + maxThreadsSize - 1) / maxThreadsSize, dimBlock>>>(vol.data, chunk_mean, chunk_std, 10, volsize);
        int j = 0;
        for (j = 0; j + 20 < thickness; j += 20)
            mrcvol.WriteBlock<unsigned char>(j, j + 20, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
        mrcvol.WriteBlock<unsigned char>(j, thickness, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
    }
    else
    {
        int j = 0;
        for (j = 0; j + 5 < thickness; j += 5)
            mrcvol.WriteBlock<float>(j, j + 5, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        mrcvol.WriteBlock<float>(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
    }
    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipFree(valvol);
    hipFree(wtvol);
    hipFree(proj.data);
}

void CuSARTZ(Point3DF &origin, MrcStackM &projs, std::vector<SimCoeff> &params,
             int thickness, MrcStackM &mrcvol, Slice &proj, Volume &vol,
             int iteration,
             float gamma, const options &opt)
{
    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;

    int batchsize = 1;
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * thickness;
    size_t projsize = (size_t)projs.X() * projs.Y() * batchsize;
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR
    CuTaskDataZ cudev;
    CuMallocSIRTTaskDataZ(cudev, projs.Z(), projs.X(), projs.Y(), thickness, batchsize);
    CUERR
    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(),
               hipMemcpyHostToDevice);
    CUERR

    dim3 dimBlock = maxThreadsSize;
    dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  thickness, batchsize);
    dim3 dim2Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  batchsize);

    CUERR
    hipMallocManaged((void **)&proj.data, sizeof(float) * (size_t)projs.X() * projs.Y() * projs.Z());
    {
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
    }
    for (int iter = 0; iter < iteration; ++iter)
    {
        for (int projIdxStart = 0; projIdxStart < projsnum; projIdxStart += batchsize)
        {
            float *curProjData = proj.data + projIdxStart * projs.X() * projs.Y();
            hipDeviceSynchronize();
            printf("SART Iter %d on projs [%d,%d)\n", iter, projIdxStart, projIdxStart + batchsize);
            hipMemset(cudev.c, 0, sizeof(float) * projsize);
            hipMemset(cudev.s, 0, sizeof(float) * projsize);
            CuReprojectKernelZ<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs,
                                                       vol.data, cudev.s, cudev.c,
                                                       cudev.x, cudev.y, 0, projIdxStart);
            CUERR
            hipDeviceSynchronize();
            CuCalcProjectionDiffKernelZ<<<dim2Grid, dimBlock>>>(
                curProjData, cudev.s, cudev.c, cudev.x, cudev.y);
            CUERR
            hipDeviceSynchronize();
            CuUpdateVolumeByProjDiffKernelZ<<<dim3Grid, dimBlock>>>(
                cudev.origin, cudev.coeffs, vol.data, cudev.s, gamma, cudev.x, cudev.y,
                0, projIdxStart);
            hipDeviceSynchronize();
            CUERR
        }
    }
    if (opt.f2b)
    {
        thrust::device_ptr<float> dev_ptr(vol.data);
        float chunk_mean = thrust::reduce(dev_ptr, dev_ptr + volsize) / volsize;
        float chunk_std = thrust::transform_reduce(dev_ptr, dev_ptr + volsize, [chunk_mean] __device__(float x)
                                                   { return (x - chunk_mean) * (x - chunk_mean); }, 0.0f, thrust::plus<float>());
        chunk_std = sqrt(chunk_std / volsize);

        CufloatToByteKernel<<<(volsize / 4 + maxThreadsSize - 1) / maxThreadsSize, dimBlock>>>(vol.data, chunk_mean, chunk_std, 10, volsize);
        int j = 0;
        for (j = 0; j + 20 < thickness; j += 20)
            mrcvol.WriteBlock<unsigned char>(j, j + 20, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
        mrcvol.WriteBlock<unsigned char>(j, thickness, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
    }
    else
        {
        int j = 0;
        for (j = 0; j + 5 < thickness; j += 5)
            mrcvol.WriteBlock<float>(j, j + 5, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        mrcvol.WriteBlock<float>(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
    }

    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipHostFree(proj.data);
}

void CuFBPZ(Point3DF &origin, MrcStackM &projs,
            std::vector<SimCoeff> &params, int thickness,
            MrcStackM &mrcvol, Slice &proj,
            Volume &vol, int filterMode, const options &opt)
{
    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;

    int steplength = thickness;
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * steplength;
    size_t projsize = (size_t)projs.X() * projs.Y() * projsnum;
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR
    CuTaskDataZ cudev;
    CuMallocBPTTaskDataZ(cudev, projsnum, projs.X(), projs.Y(), steplength);
    float *originalProjsData;
    hipMallocManaged((void **)(&originalProjsData), sizeof(float) * projsize); // the data of proj is extended by steplength
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(),
               hipMemcpyHostToDevice);
    CUERR
    {
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (originalProjsData + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (originalProjsData + (size_t)projs.X() * projs.Y() * j));
    }
    size_t ny = projs.header.ny;

    ApplyFilterInplace(projs, originalProjsData, ny, filterMode);

    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR
    for (int z = vol.z; z < vol.z + vol.height; z += steplength)
    {
        if (z + steplength >= vol.z + vol.height)
        { // compenstate for margin
            steplength = vol.z + vol.height - z;
            volsize = (size_t)projs.X() * projs.Y() * steplength;
            projsize = (size_t)projs.X() * projs.Y() * projsnum;
            cudev.z = steplength;
        }

        hipDeviceSynchronize();
        CUERR
        dim3 dimBlock = maxThreadsSize;
        dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                      steplength, projsnum);

        CuBackProjKernelZ<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs,
                                                  vol.data, originalProjsData, cudev.x,
                                                  cudev.y, z);
        CUERR
        hipDeviceSynchronize();
        CUERR
             if (opt.f2b)
        {
            thrust::device_ptr<float> dev_ptr(vol.data);
            float chunk_mean = thrust::reduce(dev_ptr, dev_ptr + volsize) / volsize;
            float chunk_std = thrust::transform_reduce(dev_ptr, dev_ptr + volsize, [chunk_mean] __device__(float x)
                                                       { return (x - chunk_mean) * (x - chunk_mean); }, 0.0f, thrust::plus<float>());
            chunk_std = sqrt(chunk_std / volsize);

            CufloatToByteKernel<<<(volsize / 4 + maxThreadsSize - 1) / maxThreadsSize, dimBlock>>>(vol.data, chunk_mean, chunk_std, 10, volsize);
            int j = 0;
            for (j = 0; j + 20 < thickness; j += 20)
                mrcvol.WriteBlock<unsigned char>(j, j + 20, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
            mrcvol.WriteBlock<unsigned char>(j, thickness, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
        }
        else
        {
            int j = 0;
            for (j = 0; j + 20 < thickness; j += 20)
                mrcvol.WriteBlock<float>(j, j + 20, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
            mrcvol.WriteBlock<float>(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        }
    }
    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipFree(originalProjsData);
}

void CuADMMZ(Point3DF &origin, MrcStackM &projs, std::vector<SimCoeff> &params,
             int thickness, MrcStackM &mrcvol, Slice &proj, Volume &vol,
             int iteration, int cgiter, float gamma, float soft, const options &opt)
{
    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;
    int batchsize = 1;
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * thickness;
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR

    CuTaskDataZ cudev;
    CuMallocADMMTaskDataZ(cudev, projs.Z(), projs.X(), projs.Y(), thickness, batchsize);
    CUERR
    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(), hipMemcpyHostToDevice);
    CUERR

    dim3 dimBlock = maxThreadsSize;
    dim3 dim1Grid((volsize + maxThreadsSize - 1) / maxThreadsSize);
    dim3 dim2Grid_xy((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize, thickness);
    dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  thickness, batchsize);
    dim3 dim3Grid_xyz((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                      thickness, projs.Z());
    CUERR

    hipMallocManaged((void **)&proj.data, sizeof(float) * (size_t)projs.X() * projs.Y() * projs.Z());
    {
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
    }



    float *htb, *uk, *dk;
    hipMallocManaged((void **)&htb, sizeof(float) * volsize);
    CUERR
    hipMallocManaged((void **)&uk, sizeof(float) * volsize);
    CUERR
    hipMallocManaged((void **)&dk, sizeof(float) * volsize);
    CUERR
    
    hipMemset(htb, 0, sizeof(float) * volsize);
    hipMemset(uk, 0, sizeof(float) * volsize);
    hipMemset(dk, 0, sizeof(float) * volsize);

   for (int projIdxStart = 0; projIdxStart < projsnum; projIdxStart += batchsize) 
    {
        float *curProjData = proj.data + projIdxStart * projs.X() * projs.Y();
        hipDeviceSynchronize();

        CuAtb_ADMM_Z<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs, htb, cudev.x, cudev.y, 
                                            curProjData, 0, projIdxStart);
        hipDeviceSynchronize();
        CUERR
    }

    for (int iter = 0; iter < iteration; ++iter)
    {
        float *x0;
        hipMallocManaged((void **)&x0, sizeof(float) * volsize);
        CUERR
        hipMemset(x0, 0, sizeof(float) * volsize);

        printf("ADMM Iter %d \n", iter);                               
            
        CuATbGammaIt_ADMM_Z<<<dim1Grid, dimBlock>>>(htb, uk, dk, volsize, gamma);
        hipDeviceSynchronize();
        CUERR

        CuATaGammaI_ADMM_Z(cudev.origin, cudev.coeffs, cudev.s, cudev.c, x0, cudev.x, cudev.y, 
                               volsize, vol.data, gamma, dim1Grid, dim3Grid, dimBlock, 0, projsnum); 
        CUERR
        hipDeviceSynchronize();
        CUERR

        CuApplycg_ADMM_Z(cudev, vol.data, x0, htb, cgiter, gamma, volsize, dim1Grid, dim3Grid, dimBlock, projsnum);
        CUERR
        hipFree(x0);

        //update uk
        CuSoft_ADMM_Z<<<dim1Grid, dimBlock>>>(uk, dk, vol.data, soft, volsize);
        CUERR
        hipDeviceSynchronize();

        //update dk
        Cu_dk_ADMM_Z<<<dim1Grid, dimBlock>>>(dk, uk, vol.data, volsize);
        CUERR
        hipDeviceSynchronize();

        CUERR
    }
    hipDeviceSynchronize();
    CUERR

    
    if (opt.f2b)
    {
        thrust::device_ptr<float> dev_ptr(vol.data);
        float chunk_mean = thrust::reduce(dev_ptr, dev_ptr + volsize) / volsize;
        float chunk_std = thrust::transform_reduce(dev_ptr, dev_ptr + volsize, [chunk_mean] __device__(float x)
                                                   { return (x - chunk_mean) * (x - chunk_mean); }, 0.0f, thrust::plus<float>());
        chunk_std = sqrt(chunk_std / volsize);

        CufloatToByteKernel<<<(volsize / 4 + maxThreadsSize - 1) / maxThreadsSize, dimBlock>>>(vol.data, chunk_mean, chunk_std, 10, volsize);
        int j = 0;
        for (j = 0; j + 20 < thickness; j += 20)
            mrcvol.WriteBlock<unsigned char>(j, j + 20, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
        mrcvol.WriteBlock<unsigned char>(j, thickness, 'z', (vol.data + ((size_t)projs.X() * projs.Y() * j) / 4));
    }
    else
    {
        int j = 0;
        for (j = 0; j + 5 < thickness; j += 5)
            mrcvol.WriteBlock<float>(j, j + 5, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        mrcvol.WriteBlock<float>(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
    }


    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipFree(htb);
    hipFree(uk);
    hipFree(dk);
    hipFree(proj.data);
}

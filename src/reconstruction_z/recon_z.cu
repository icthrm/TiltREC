#include "recon_z.cuh"
extern hipDeviceProp_t deviceProps;

void CuBackProjectZ(Point3DF &origin, MrcStackM &projs,
                    std::vector<SimCoeff> &params, int thickness,
                    MrcStackM &mrcvol, Slice &proj,
                    Volume &vol)
{

    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;

    int steplength = thickness; 
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * steplength;
    size_t projsize = (size_t)projs.X() * projs.Y() * projsnum; 
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR
    CuTaskDataZ cudev;
    CuMallocBPTTaskDataZ(cudev, projsnum, projs.X(), projs.Y(), steplength);
    float *originalProjsData;
    hipMallocManaged((void **)(&originalProjsData), sizeof(float) * projsize); // the data of proj is extended by steplength
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(),
               hipMemcpyHostToDevice);
    CUERR
    { 
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (originalProjsData + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (originalProjsData + (size_t)projs.X() * projs.Y() * j));
    }
    CUERR
    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR

    for (int z = vol.z; z < vol.z + vol.height; z += steplength)
    {
        std::cout << "BPT reconstructs " << z  << std::endl;
        if (z + steplength >= vol.z + vol.height)
        { // compenstate for margin
            steplength = vol.z + vol.height - z;
            volsize = (size_t)projs.X() * projs.Y() * steplength;
            projsize = (size_t)projs.X() * projs.Y() * projsnum;
            cudev.z = steplength;
        }

        time_t now = time(NULL);
        char *curr_time = ctime(&now);
        std::cout << curr_time << std::endl;

        hipDeviceSynchronize();
        CUERR
        dim3 dimBlock = maxThreadsSize;
        dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                      steplength, projsnum);
        CuBackProjKernelZ<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs,
                                                  vol.data, originalProjsData, cudev.x,
                                                  cudev.y, z);
        CUERR
        hipDeviceSynchronize();
        CUERR
        printf("Write data to file\n");
        {
            int j = 0;
            for (j = 0; j + 20 < thickness; j += 20)
                mrcvol.WriteBlock(j, j + 20, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
            mrcvol.WriteBlock(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        }
    }
    time_t now = time(NULL);
    char *curr_time = ctime(&now);
    std::cout << curr_time << std::endl;
    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipFree(originalProjsData);
}

void CuSIRTZ(Point3DF &origin, MrcStackM &projs, std::vector<SimCoeff> &params,
             int thickness, MrcStackM &mrcvol, Slice &proj, Volume &vol,
             int iteration,
             float gamma)
{

    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;

    int batchsize = 1; 
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * thickness;
    size_t oneProjsize = (size_t)projs.X() * projs.Y() * batchsize; 
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR
    CuTaskDataZ cudev;
    CuMallocSIRTTaskDataZ(cudev, projs.Z(), projs.X(), projs.Y(), thickness, batchsize);
    CUERR
    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(),
               hipMemcpyHostToDevice);
    CUERR

    float *valvol, *wtvol;
    dim3 dimBlock = maxThreadsSize;
    dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  thickness, batchsize);
    dim3 dim2Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  batchsize);
    hipMallocManaged((void **)&valvol, sizeof(float) * volsize);
    hipMallocManaged((void **)&wtvol, sizeof(float) * volsize);
    CUERR
    hipMallocManaged((void **)&proj.data, sizeof(float) * (size_t)projs.X() * projs.Y() * projs.Z());
    { 
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
    }

    for (int iter = 0; iter < iteration; ++iter)
    {
        hipMemset(valvol, 0, sizeof(float) * volsize);
        hipMemset(wtvol, 0, sizeof(float) * volsize);
        for (int projIdxStart = 0; projIdxStart < projsnum; projIdxStart += batchsize) 
        {
            float *curProjData = proj.data + projIdxStart * projs.X() * projs.Y();
            hipDeviceSynchronize();
            printf("SIRT Iter %d on projs [%d,%d)\n", iter, projIdxStart, projIdxStart + batchsize);
            hipMemset(cudev.c, 0, sizeof(float) * oneProjsize);
            hipMemset(cudev.s, 0, sizeof(float) * oneProjsize);
            CuReprojectKernelZ<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs,
                                                       vol.data, cudev.s, cudev.c,
                                                       cudev.x, cudev.y, 0, projIdxStart); 
            CUERR

            CuCalcProjectionDiffKernelZ<<<dim2Grid, dimBlock>>>(
                curProjData, cudev.s, cudev.c, cudev.x, cudev.y);
            CUERR
            CuBackProjWeightAndValueKernelZ<<<dim3Grid, dimBlock>>>(
                cudev.origin, cudev.coeffs, valvol, wtvol, cudev.s, cudev.x, cudev.y,
                0, projIdxStart);

            hipDeviceSynchronize();

            CUERR
        }

        CuUpdateVolumeByWeightsKernelZ<<<
            (volsize + maxThreadsSize - 1) / maxThreadsSize, dimBlock>>>(
            vol.data, valvol, wtvol, gamma, volsize);
        CUERR
        hipDeviceSynchronize();
        CUERR
    }
    hipDeviceSynchronize();
    CUERR

    printf("Writing data to file\n");
    {
        int j = 0;
        for (j = 0; j + 5 < thickness; j += 5)
            mrcvol.WriteBlock(j, j + 5, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        mrcvol.WriteBlock(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
    }
    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipFree(valvol);
    hipFree(wtvol);
    hipFree(proj.data);
}

void CuSARTZ(Point3DF &origin, MrcStackM &projs, std::vector<SimCoeff> &params,
             int thickness, MrcStackM &mrcvol, Slice &proj, Volume &vol,
             int iteration,
             float gamma)
{
    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;

    int batchsize = 1; 
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * thickness;
    size_t projsize = (size_t)projs.X() * projs.Y() * batchsize;
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR
    CuTaskDataZ cudev;
    CuMallocSIRTTaskDataZ(cudev, projs.Z(), projs.X(), projs.Y(), thickness, batchsize);
    CUERR
    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(),
               hipMemcpyHostToDevice);
    CUERR

    dim3 dimBlock = maxThreadsSize;
    dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  thickness, batchsize);
    dim3 dim2Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  batchsize);

    CUERR
    hipMallocManaged((void **)&proj.data, sizeof(float) * (size_t)projs.X() * projs.Y() * projs.Z());
    { 
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
    }
    for (int iter = 0; iter < iteration; ++iter)
    {
        for (int projIdxStart = 0; projIdxStart < projsnum; projIdxStart += batchsize) 
        {
            float *curProjData = proj.data + projIdxStart * projs.X() * projs.Y();
            hipDeviceSynchronize();
            printf("SART Iter %d on projs [%d,%d)\n", iter, projIdxStart, projIdxStart + batchsize);
            hipMemset(cudev.c, 0, sizeof(float) * projsize);
            hipMemset(cudev.s, 0, sizeof(float) * projsize);
            CuReprojectKernelZ<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs,
                                                       vol.data, cudev.s, cudev.c,
                                                       cudev.x, cudev.y, 0, projIdxStart); 
            CUERR
            hipDeviceSynchronize();
            CuCalcProjectionDiffKernelZ<<<dim2Grid, dimBlock>>>(
                curProjData, cudev.s, cudev.c, cudev.x, cudev.y);
            CUERR
            hipDeviceSynchronize();
            CuUpdateVolumeByProjDiffKernelZ<<<dim3Grid, dimBlock>>>(
                cudev.origin, cudev.coeffs, vol.data, cudev.s, gamma, cudev.x, cudev.y,
                0, projIdxStart);
            hipDeviceSynchronize();
            CUERR
        }
    }
    {
        printf("Write data to file\n");
        int j = 0;
        for (j = 0; j + 5 < thickness; j += 5)
            mrcvol.WriteBlock(j, j + 5, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        mrcvol.WriteBlock(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
    }

    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipHostFree(proj.data);
}

void CuFBPZ(Point3DF &origin, MrcStackM &projs,
           std::vector<SimCoeff> &params, int thickness,
           MrcStackM &mrcvol, Slice &proj,
           Volume &vol, int filterMode)
{
    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;

    int steplength = thickness; 
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * steplength;
    size_t projsize = (size_t)projs.X() * projs.Y() * projsnum; 
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR
    CuTaskDataZ cudev;
    CuMallocBPTTaskDataZ(cudev, projsnum, projs.X(), projs.Y(), steplength);
    float *originalProjsData;
    hipMallocManaged((void **)(&originalProjsData), sizeof(float) * projsize); // the data of proj is extended by steplength
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(),
               hipMemcpyHostToDevice);
    CUERR
    { 
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (originalProjsData + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (originalProjsData + (size_t)projs.X() * projs.Y() * j));
    }
    size_t ny = projs.header.ny;

    ApplyFilterInplace(projs, originalProjsData, ny, filterMode); 

    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice);
    CUERR
    for (int z = vol.z; z < vol.z + vol.height; z += steplength)
    {
        if (z + steplength >= vol.z + vol.height)
        { // compenstate for margin
            steplength = vol.z + vol.height - z;
            volsize = (size_t)projs.X() * projs.Y() * steplength;
            projsize = (size_t)projs.X() * projs.Y() * projsnum;
            cudev.z = steplength;
        }

        time_t now = time(NULL);
        char *curr_time = ctime(&now);
        std::cout << curr_time << std::endl;

        hipDeviceSynchronize();
        CUERR
        dim3 dimBlock = maxThreadsSize;
        dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                      steplength, projsnum);

        CuBackProjKernelZ<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs,
                                                  vol.data, originalProjsData, cudev.x,
                                                  cudev.y, z);
        CUERR
        hipDeviceSynchronize();
        CUERR
        printf("Write data to file\n");
        {
            int j = 0;
            for (j = 0; j + 20 < thickness; j += 20)
                mrcvol.WriteBlock(j, j + 20, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
            mrcvol.WriteBlock(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        }
    }
    time_t now = time(NULL);
    char *curr_time = ctime(&now);
    std::cout << curr_time << std::endl;
    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipFree(originalProjsData);
}

void CuADMMZ(Point3DF &origin, MrcStackM &projs, std::vector<SimCoeff> &params,
             int thickness, MrcStackM &mrcvol, Slice &proj, Volume &vol,
             int iteration, int cgiter, float gamma, float soft)
{
    size_t maxThreadsSize = deviceProps.maxThreadsPerBlock;
    int batchsize = 1; 
    int projsnum = projs.Z();
    size_t volsize = (size_t)projs.X() * projs.Y() * thickness; 
    hipMallocManaged((void **)(&vol.data), sizeof(float) * volsize);
    CUERR

    CuTaskDataZ cudev;
    CuMallocADMMTaskDataZ(cudev, projs.Z(), projs.X(), projs.Y(), thickness, batchsize);
    CUERR
    hipMemcpy(cudev.origin, &origin, sizeof(Point3DF), hipMemcpyHostToDevice); 
    CUERR
    hipMemcpy(cudev.coeffs, &(params[0]), sizeof(SimCoeff) * params.size(), hipMemcpyHostToDevice);
    CUERR

    dim3 dimBlock = maxThreadsSize;
    dim3 dim1Grid((volsize + maxThreadsSize - 1) / maxThreadsSize);  
    dim3 dim2Grid_xy((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize, thickness);
    dim3 dim3Grid((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  thickness, batchsize);
    dim3 dim3Grid_xyz((projs.X() * projs.Y() + maxThreadsSize - 1) / maxThreadsSize,
                  thickness, projs.Z());
    CUERR
    
    hipMallocManaged((void **)&proj.data, sizeof(float) * (size_t)projs.X() * projs.Y() * projs.Z());
    { 
        int j = 0;
        for (j = 0; j + 5 < projs.Z(); j += 5)
            projs.ReadBlock(j, j + 5, 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
        projs.ReadBlock(j, projs.Z(), 'z', (proj.data + (size_t)projs.X() * projs.Y() * j));
    }

    hipEvent_t begin, stop;
    hipEventCreate(&begin);
    hipEventCreate(&stop);
    hipEventRecord(begin);
    std::cout << "ADMM reconstruction started." << std::endl;

    float *htb, *x0, *uk, *dk;
    hipMallocManaged((void **)&htb, sizeof(float) * volsize);
    CUERR
    hipMallocManaged((void **)&x0, sizeof(float) * volsize);
    CUERR
    hipMallocManaged((void **)&uk, sizeof(float) * volsize);
    CUERR
    hipMallocManaged((void **)&dk, sizeof(float) * volsize);
    CUERR
    
    hipMemset(uk, 0, sizeof(float) * volsize);
    hipMemset(dk, 0, sizeof(float) * volsize);

    for (int iter = 0; iter < iteration; ++iter)
    {
        hipMemset(htb, 0, sizeof(float) * volsize);
        hipMemset(x0, 0, sizeof(float) * volsize);

        for (int projIdxStart = 0; projIdxStart < projsnum; projIdxStart += batchsize) 
        {
            float *curProjData = proj.data + projIdxStart * projs.X() * projs.Y();
            hipDeviceSynchronize();
            printf("ADMM Iter %d on projs [%d,%d)\n", iter, projIdxStart, projIdxStart + batchsize);

            CuAtb_ADMM_Z<<<dim3Grid, dimBlock>>>(cudev.origin, cudev.coeffs, htb, cudev.x, cudev.y, 
                                                 curProjData, 0, projIdxStart);
            hipDeviceSynchronize();                                     
            CUERR
            CuATbGammaIt_ADMM_Z<<<dim1Grid, dimBlock>>>(htb, uk, dk, volsize, gamma);
            hipDeviceSynchronize();
            CUERR
            CuATaGammaI_ADMM_Z(cudev.origin, cudev.coeffs, cudev.s, cudev.c, x0, cudev.x, cudev.y, 
                               volsize, vol.data, gamma, dim1Grid, dim3Grid, dimBlock, 0, projIdxStart); 
            CUERR
            hipDeviceSynchronize();
        }
        CuApplycg_ADMM_Z(cudev, vol.data, x0, htb, cgiter, gamma, volsize, dim1Grid, dim3Grid_xyz, dimBlock);
        CUERR
        CuSoft_ADMM_Z<<<dim1Grid, dimBlock>>>(uk, dk, soft, cudev.x, volsize);
        CUERR
        hipDeviceSynchronize();
        CUERR
    }
    hipDeviceSynchronize();
    CUERR

    printf("Writing data to file\n");
    {
        int j = 0;
        for (j = 0; j + 5 < thickness; j += 5)
            mrcvol.WriteBlock(j, j + 5, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
        mrcvol.WriteBlock(j, thickness, 'z', (vol.data + (size_t)projs.X() * projs.Y() * j));
    }
    CHECK_CUDA(hipEventRecord(stop))
    CHECK_CUDA(hipEventSynchronize(stop))
    float milliseconds = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, begin, stop))
    std::cout << "ADMM reconstruction completed." << std::endl;
    std::cout << "Reconstruction time: " << milliseconds / 1000.0f << "s." << std::endl;
    CHECK_CUDA(hipEventDestroy(begin))
    CHECK_CUDA(hipEventDestroy(stop))
    
    hipFree(vol.data);
    CuFreeTaskDataZ(cudev);
    hipFree(htb);
    hipFree(x0);
    hipFree(uk);
    hipFree(dk);
    hipFree(proj.data);
}

